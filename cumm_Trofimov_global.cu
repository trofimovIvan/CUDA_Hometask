#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <random>

#define BLOCK_SIZE 32
#define N 2048

__global__ void MatrixMultiplication_global (double *dA, double *dB, double *dC){
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	double sum = 0;
	int ia = N*(BLOCK_SIZE * by + ty); // номер строки А
	int ib = BLOCK_SIZE*bx + tx; // номер столбца В
	int ic = ia + ib; // номер элемента С
	// вычисление элемента С
	for (int k=0; k<N; k++) sum += dA[ia + k]*dB[ib + k*N];
	dC[ic] = sum;
}

int main()
{
	int numBytes = N*N*sizeof(double);
	double *dA, *dB, *dC, *hA, *hB, *hC;
	// задание сетки нитей и блоков:
	dim3 threads (BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks (N/threads.x, N/threads.y);	
	hipEvent_t start, stop;
	hipEventCreate( &start);
	hipEventCreate( &stop);
	float time;

	//выделение памяти на GPU:
	hipMalloc( (void**) &dA, numBytes );
	hipMalloc( (void**) &dB, numBytes );
	hipMalloc( (void**) &dC, numBytes );

	//выделение памяти на HOST и заполнение матриц
	hA = (double*) malloc(numBytes);
	hB = (double*) malloc(numBytes);
	hC = (double*) malloc(numBytes);
	
	for (int i = 0; i< N; i++){
		for (int j = 0; j < N; j++){
			hA[j + i*N] = (double) rand() / RAND_MAX;
			hB[j + i*N] = (double) rand() / RAND_MAX;
			hC[j + i*N] = 0;
			}
	}
	//копирование матриц на GPU
	hipMemcpy(dA, hA, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, numBytes, hipMemcpyHostToDevice);
	// Засекаем время и умножаем
	hipEventRecord (start, 0);
	MatrixMultiplication_global <<<blocks, threads >>> (dA, dB, dC);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime (&time, start, stop);
	printf("GPU time = %f ms\n ",time);

	//копирование с GPU на HOST
	hipMemcpy(hC, dC, numBytes, hipMemcpyDeviceToHost);

	//Освобождение памяти
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	free(hA);
	free(hB);
	free(hC);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	return 0;
}
