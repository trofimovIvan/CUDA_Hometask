#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <random>

#define BLOCK_SIZE 32
#define N 2048

__global__ void MatrixMultiplication_global (double *dA, double *dB, double *dC){
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	double sum = 0;
	int ia = N*(BLOCK_SIZE * by + ty); // номер строки А
	int ib = BLOCK_SIZE*bx + tx; // номер столбца В
	int ic = ia + ib; // номер элемента С
	// вычисление элемента С
	for (int k=0; k<N; k++) sum += dA[ia + k]*dB[ib + k*N];
	dC[ic] = sum;
}

int main()
{
	int numBytes = N*N*sizeof(double);
	double *dA, *dB, *dC, *hA, *hB, *hC;
	// задание сетки нитей и блоков:
	dim3 threads (BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks (N/threads.x, N/threads.y);	
	hipEvent_t start, stop;
	hipEventCreate( &start);
	hipEventCreate( &stop);
	float time;

	//выделение памяти на GPU:
	hipMalloc( (void**) &dA, numBytes );
	hipMalloc( (void**) &dB, numBytes );
	hipMalloc( (void**) &dC, numBytes );

	//выделение памяти на HOST и заполнение матриц pinned
	hipHostAlloc( (void**) &hA, numBytes, hipHostMallocDefault);	
	hipHostAlloc( (void**) &hB, numBytes, hipHostMallocDefault);
	hipHostAlloc( (void**) &hC, numBytes, hipHostMallocDefault);
	
	for (int i = 0; i< N; i++){
		for (int j = 0; j < N; j++){
			hA[j + i*N] = (double) rand() / RAND_MAX;
			hB[j + i*N] = (double) rand() / RAND_MAX;
			hC[j + i*N] = 0;
			}
	}

	//копирование матриц на GPU
	hipEventRecord(start, 0);
	hipMemcpy(dA, hA, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, numBytes, hipMemcpyHostToDevice);
	// Засекаем время и умножаем
	MatrixMultiplication_global <<<blocks, threads >>> (dA, dB, dC);
	hipDeviceSynchronize();
	hipEventSynchronize(stop);

	//копирование с GPU на HOST
	hipMemcpy(hC, dC, numBytes, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime (&time, start, stop);
	printf("GPU time pinned = %f ms \n", time);
	//Освобождение памяти
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	hipFree(hA);
	hipFree(hB);
	hipFree(hC);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}	
	
