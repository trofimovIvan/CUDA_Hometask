
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <random>

#define BLOCK_SIZE 32
#define N 2048
#define NSTREAM 2

__global__ void MatrixMultiplication_global (double *dA, double *dB, double *dC){
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	double sum = 0;
	int ia = N*(BLOCK_SIZE * by + ty); // номер строки А
	int ib = BLOCK_SIZE*bx + tx; // номер столбца В
	int ic = ia + ib; // номер элемента С
	// вычисление элемента С
	for (int k=0; k<N; k++) sum += dA[ia + k]*dB[ib + k*N];
	dC[ic] = sum;
}

int main()
{
	int numBytes = N*N*sizeof(double);
	double *dA, *dB, *dC, *hA, *hB, *hC;
	// задание сетки нитей и блоков:
	dim3 threads (BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks (N/threads.x, N/threads.y);	
	hipEvent_t start, stop;
	dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);

        dim3 gridDim(ceilf(N/(float)BLOCK_SIZE), ceilf(N/(float)BLOCK_SIZE), 1);
	hipEventCreate( &start);
	hipEventCreate( &stop);
	float time;
	//создаем cuda-потоки
	hipStream_t stream[NSTREAM];
	for (int i =0; i<NSTREAM; i++) hipStreamCreate(&stream[i]);

   	dim3 gridDim_s(ceilf(N/(float)BLOCK_SIZE/NSTREAM));
	//выделение памяти на GPU:
	hipMalloc( (void**) &dA, numBytes );
	hipMalloc( (void**) &dB, numBytes );
	hipMalloc( (void**) &dC, numBytes );

	//выделение памяти на HOST и заполнение матриц pinned
	hipHostMalloc( (void**) &hA, numBytes, hipHostMallocDefault);	
	hipHostMalloc( (void**) &hB, numBytes, hipHostMallocDefault);
	hipHostMalloc( (void**) &hC, numBytes, hipHostMallocDefault);
	
	for (int i = 0; i< N; i++){
		for (int j = 0; j < N; j++){
			hA[j + i*N] = (double) rand() / RAND_MAX;
			hB[j + i*N] = (double) rand() / RAND_MAX;
			hC[j + i*N] = 0;
			}
	}
	//копирование матриц на GPU
	hipEventRecord(start, 0);
	
    for (int i = 0; i < NSTREAM; ++i) {
	    hipMemcpyAsync(
                dA + (int)i * N / NSTREAM,
                hA + (int)i * N / NSTREAM,
                sizeof(double) * N / NSTREAM,
                hipMemcpyHostToDevice,
                stream[i]);
        
        hipMemcpyAsync(
                dB + (int)i * N / NSTREAM,
                hB + (int)i * N / NSTREAM,
                sizeof(double) * N / NSTREAM,
                hipMemcpyHostToDevice,
                stream[i]);
    }
    for (int i = 0; i < NSTREAM; ++i) {
	    MatrixMultiplication_global <<<gridDim_s, blockDim, 0, stream[i]>>>
                (dA + (int)i * N / NSTREAM, dB + (int)i * N / NSTREAM,
                 dC + (int)i * N / NSTREAM);
    }
    for (int i = 0; i < NSTREAM; ++i) {
	    hipMemcpyAsync(
                hC + (int)i * N / NSTREAM, 
                dC + (int)i * N / NSTREAM,
                sizeof(double) * N / NSTREAM,
                hipMemcpyDeviceToHost,
                stream[i]) << '\n';
    }
    hipDeviceSynchronize();
    for (int i = 0; i < NSTREAM; ++i) hipStreamDestroy(stream[i]);
    
 
	//cudaDeviceSynchronize();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime (&time, start, stop);
	printf("GPU time pinned async = %3.1f ms \n", time);
	//Освобождение памяти
	for (int i =0; i<NSTREAM; i++) hipStreamDestroy(stream[i]);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	hipFree(hA);
	hipFree(hB);
	hipFree(hC);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}	
	
