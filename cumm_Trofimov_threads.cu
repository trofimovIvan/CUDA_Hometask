#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <random>

#define BLOCK_SIZE 32
#define N 2048

__global__ void MatrixMultiplication_global (double *dA, double *dB, double *dC){
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	double sum = 0;
	int ia = N*(BLOCK_SIZE * by + ty); // номер строки А
	int ib = BLOCK_SIZE*bx + tx; // номер столбца В
	int ic = ia + ib; // номер элемента С
	// вычисление элемента С
	for (int k=0; k<N; k++) sum += dA[ia + k]*dB[ib + k*N];
	dC[ic] = sum;
}

int main()
{
	int numBytes = N*N*sizeof(double);
	double *dA, *dB, *dC, *hA, *hB, *hC;
	// задание сетки нитей и блоков:
	dim3 threads (BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks (N/threads.x, N/threads.y);	
	hipEvent_t start, stop;
	hipEventCreate( &start);
	hipEventCreate( &stop);
	float time;
	//создаем cuda-потоки
	hipStream_t stream[2];
	for (int i =0; i<2; i++) hipStreamCreate(&stream[i]);

	//выделение памяти на GPU:
	hipMalloc( (void**) &dA, numBytes );
	hipMalloc( (void**) &dB, numBytes );
	hipMalloc( (void**) &dC, numBytes );

	//выделение памяти на HOST и заполнение матриц pinned
	hipHostMalloc( (void**) &hA,2*numBytes);	
	hipHostMalloc( (void**) &hB, 2*numBytes);
	hipHostMalloc( (void**) &hC, 2*numBytes);
	
	for (int i = 0; i< N; i++){
		for (int j = 0; j < N; j++){
			hA[j + i*N] = (double) rand() / RAND_MAX;
			hB[j + i*N] = (double) rand() / RAND_MAX;
			hC[j + i*N] = 0;
			}
	}
	//копирование матриц на GPU
	hipEventRecord(start, 0);
	for (int i = 0; i<2; i++) {
		hipMemcpyAsync(dA, hA, numBytes, hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(dB, hB, numBytes, hipMemcpyHostToDevice, stream[i]);
	}
	for (int i = 0; i<2; i++){

	MatrixMultiplication_global <<<blocks, threads, 0, stream[i] >>> (dA, dB, dC);
	hipDeviceSynchronize();
	}
	//копирование с GPU на HOST
	for (int i = 0; i<2; i++){
		hipMemcpyAsync(hC, dC, numBytes, hipMemcpyDeviceToHost, stream[i]);
	}
	hipDeviceSynchronize();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime (&time, start, stop);
	printf("GPU time pinned = %f ms \n", time);
	//Освобождение памяти
	for (int i =0; i<2; i++) hipStreamDestroy(stream[i]);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	hipFree(hA);
	hipFree(hB);
	hipFree(hC);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}	
	
